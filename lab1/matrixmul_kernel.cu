#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * This software and the information contained herein is PROPRIETARY and 
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and 
 * conditions of a Non-Disclosure Agreement.  Any reproduction or 
 * disclosure to any third party without the express written consent of 
 * NVIDIA is prohibited.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* Matrix multiplication: P = M * N.
 * Device code.
 */

#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_

#include <stdio.h>
#include "matrixmul.h"

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
// Matrix multiplication kernel thread specification
__global__ void MatrixMulKernel(Matrix M, Matrix N, Matrix P)
{
	// Set up for a loop to calculate the dot product of the row threadIdx.y in M with the column threadIdx.x of N.
	// As such, each thread will calculate the element of the solution matrix P[threadIdx.y][threadIdx.x].
	// NOTE: M, N, and P are all square matrices of the same dimensions
	int width = P.width; 
	float dotProduct = 0;

	// Check that the row/col to iterate over are within bounds of the current block
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y; 
	if(col >= width || row >= width) return;

	// This thread will loop over a row of M and a column of N to calculate the corresponding element of P
	for (int k=0; k<width; k++) {
		float eleMent = M.elements[row * width + k];
		float elemeNt = N.elements[k * width + col];
		dotProduct += (eleMent * elemeNt);
	}

	// Store the result in P
	P.elements[row * width + col] = dotProduct;
}

#endif // #ifndef _MATRIXMUL_KERNEL_H_
